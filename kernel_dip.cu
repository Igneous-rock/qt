
#include <hip/hip_runtime.h>
#define BLOCK_WIDTH 32
#define BLOCK_HEIGHT 32
#define TILE_WIDTH 30
#define TILE_HEIGHT 30
#define NODATA -9999
#define FILTER_RADIUS 1

// -------------------------------------------------Neighbours access order is 			// 1 2 3
__constant__ int off_x[8] = {-1, 0, 1,1,1,0,-1,-1};		// 8    4
__constant__ int off_y[8] = {-1,-1,-1,0,1,1, 1, 0};		// 7 6 5

__global__ void kernel_ndvi( short *m_l,short *m_r,short *m_wi, const int w, const int h)
{
	int x = blockIdx.x * TILE_WIDTH  + threadIdx.x - FILTER_RADIUS;
	int y = blockIdx.y * TILE_HEIGHT + threadIdx.y - FILTER_RADIUS;
	//Clamp to the center
	x = max(FILTER_RADIUS, x);
	x = min(x, w - FILTER_RADIUS - 1);
	y = max(FILTER_RADIUS, y);
	y = min(y, h - FILTER_RADIUS - 1);

	unsigned int i_img = y * w + x;
	
	short v_l, v_r;
	float vf_ndvi;
	
	v_l = m_l[i_img];
	v_r = m_r[i_img];

	if (v_l != NODATA && v_r != NODATA)
	{
		vf_ndvi = __int2float_rn ( v_l - v_r ) / __int2float_rn ( v_l + v_r );
		m_wi[i_img] = __float2int_rz	(vf_ndvi * 1000.0) + 1000;
	}else{
		m_wi[i_img] = NODATA;
	}

}


__global__ void kernel_sobel( short *m_wi, float *m_sobel, const int w, const int h)
{
	__shared__ short sh_wi[BLOCK_WIDTH * BLOCK_HEIGHT];
	
	int x = blockIdx.x * TILE_WIDTH  + threadIdx.x - FILTER_RADIUS;
	int y = blockIdx.y * TILE_HEIGHT + threadIdx.y - FILTER_RADIUS;
	//Clamp to the center
	x = max(FILTER_RADIUS, x);
	x = min(x, w - FILTER_RADIUS - 1);
	y = max(FILTER_RADIUS, y);
	y = min(y, h - FILTER_RADIUS - 1);

	unsigned int i_img = y * w + x;
	unsigned int i_sh = threadIdx.y * blockDim.y + threadIdx.x;

	sh_wi[i_sh] =  m_wi[i_img];
	__syncthreads();
	
	//m_sobel[i_img] = __int2float_rn (sh_wi[i_sh]);
	if ( threadIdx.x == 0 || threadIdx.x == BLOCK_WIDTH -1 || threadIdx.y == 0 || threadIdx.y == BLOCK_HEIGHT - 1) 
	{}else{
	float sobel_x = __int2float_rn (
	   -sh_wi[i_sh - blockDim.x - 1] + sh_wi[i_sh - blockDim.x + 1]
	   -sh_wi[i_sh - 1] * 2          + sh_wi[i_sh + 1] * 2
	   -sh_wi[i_sh + blockDim.x - 1] + sh_wi[i_sh + blockDim.x + 1] );
	   
	float sobel_y = __int2float_rn (
		sh_wi[i_sh - blockDim.x - 1] + sh_wi[i_sh - blockDim.x] * 2 + sh_wi[i_sh - blockDim.x + 1] 
	   -sh_wi[i_sh + blockDim.x - 1] - sh_wi[i_sh + blockDim.x] * 2 - sh_wi[i_sh + blockDim.x + 1] );
	  
	m_sobel[i_img] = sqrtf ( sobel_x * sobel_x + sobel_y * sobel_y ) ;
	}
	/*   */


}





















